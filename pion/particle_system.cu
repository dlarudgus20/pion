#include "hip/hip_runtime.h"
#include <GL/glew.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <stdexcept>

#include "particle_system.cuh"

using namespace ps_cuda;

__global__ void particle_update_kernel(particle* particles, int num_particles, float dt)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_particles)
        return;

    auto& particle = particles[idx];

    glm::vec3 force = glm::vec3(0, -0.2f * particle.mass, 0);

    glm::vec3 a = force / particle.mass;
    glm::vec3 dv = a * dt;

    particle.position += (particle.velocity + dv * 0.5f) * dt;
    particle.velocity += dv;

    float y = particle.position.y - particle.radius + 1.1f;
    if (y < 0.0f)
    {
        particle.position.y = -y + particle.radius - 1.1f;
        particle.velocity.y *= -0.3f;
    }
}

void ps_cuda::initialize()
{
    if (hipSetDevice(0) != hipSuccess)
        throw std::runtime_error("failed to initialize cuda");
}

void ps_cuda::particle_update(std::vector<particle>& particles, float dt)
{
    constexpr int block = 128;
    int grid = (particles.size() + block - 1) / block;

    particle* device_particles;
    hipMalloc((void**)&device_particles, sizeof(particle) * particles.size());
    hipMemcpy(device_particles, particles.data(), sizeof(particle) * particles.size(), hipMemcpyHostToDevice);

    particle_update_kernel<<<grid, block>>>(device_particles, particles.size(), dt);

    hipMemcpy(particles.data(), device_particles, sizeof(particle) * particles.size(), hipMemcpyDeviceToHost);
    hipFree(device_particles);
}
